#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_kernel() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from GPU thread %d\n", idx);
}

int main() {
    printf("Testing CUDA on Jetson...\n");
    
    hello_kernel<<<2, 4>>>();
    hipDeviceSynchronize();
    
    printf("CUDA test complete.\n");
    return 0;
}